#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0
#include <chrono>
#include <iostream>
#include <micm/util/cuda_param.hpp>
#include <vector>

namespace micm
{
  namespace cuda
  {
    /// This is the CUDA kernel that performs LU decomposition on the device
    __global__ void DecomposeKernel(const double* d_A, double* d_L, double* d_U, 
                                    LuDecomposeConstDevice* devptr, size_t ngrids)
    {
      /// Local device variables
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      std::pair<size_t, size_t>* uik_nkj = devptr->d_uik_nkj_;
      std::pair<size_t, size_t>* lij_ujk = devptr->d_lij_ujk_;
      std::pair<size_t, size_t>* lkj_uji = devptr->d_lkj_uji_;
      std::pair<size_t, size_t>* lki_nkj = devptr->d_lki_nkj_;
      size_t niLU_size = devptr->d_niLU_size_;
      size_t do_aik_offset = 0;
      size_t aik_offset = 0;
      size_t uik_nkj_offset = 0;
      size_t lij_ujk_offset = 0;
      size_t do_aki_offset = 0;
      size_t aki_offset = 0;
      size_t lkj_uji_offset = 0;
      size_t lki_nkj_offset = 0;
      size_t uii_offset = 0;

      printf("this is tid %d\n", tid);
      if (tid < ngrids)
      {
        if (tid == 0) printf("JS: niLU_size = %d\n", niLU_size);
        // loop through every element in niLU
        for (size_t i = 0; i < niLU_size; i++)
        {
          // upper triangular matrix
          auto inLU = devptr->d_niLU_[i];
          for (size_t iU = 0; iU < inLU.second; ++iU)
          {
            if (devptr->d_do_aik_[do_aik_offset++])
            {
              size_t U_idx = uik_nkj[uik_nkj_offset].first + tid;
              size_t A_idx = devptr->d_aik_[aik_offset++] + tid;
              d_U[U_idx] = d_A[A_idx];
            }

            for (size_t ikj = 0; ikj < uik_nkj[uik_nkj_offset].second; ++ikj)
            {
              size_t U_idx_1 = uik_nkj[uik_nkj_offset].first + tid;
              size_t L_idx = lij_ujk[lij_ujk_offset].first + tid;
              size_t U_idx_2 = lij_ujk[lij_ujk_offset].second + tid;
              d_U[U_idx_1] -= d_L[L_idx] * d_U[U_idx_2];
              ++lij_ujk_offset;
            }
            ++uik_nkj_offset;
          }
          // lower triangular matrix

          d_L[lki_nkj[lki_nkj_offset++].first + tid] = 1.0;

          for (size_t iL = 0; iL < inLU.first; ++iL)
          {
            if (devptr->d_do_aki_[do_aki_offset++])
            {
              size_t L_idx = lki_nkj[lki_nkj_offset].first + tid;
              size_t A_idx = devptr->d_aki_[aki_offset++] + tid;
              d_L[L_idx] = d_A[A_idx];
            }
            for (size_t ikj = 0; ikj < lki_nkj[lki_nkj_offset].second; ++ikj)
            {
              size_t L_idx_1 = lki_nkj[lki_nkj_offset].first + tid;
              size_t L_idx_2 = lkj_uji[lkj_uji_offset].first + tid;
              size_t U_idx = lkj_uji[lkj_uji_offset].second + tid;
              d_L[L_idx_1] -= d_L[L_idx_2] * d_U[U_idx];
              ++lkj_uji_offset;
            }
            d_L[lki_nkj[lki_nkj_offset].first + tid] /= d_U[devptr->d_uii_[uii_offset] + tid];
            ++lki_nkj_offset;
            ++uii_offset;
          }
        }
      }
    }  // end of CUDA kernel

    /// This is the function that copies the constant data members
    /// of objects with the "CudaLuDecomposition" type to the device
    void CopyConstData(LuDecomposeConstHost* hostptr, LuDecomposeConstDevice* devptr)
    {
      /// calculate the memory space of each constant data member
      size_t niLU_size = sizeof(std::pair<size_t, size_t>) * hostptr->niLU_size_;
      size_t do_aik_size = sizeof(char) * hostptr->do_aik_size_; 
      size_t aik_size = sizeof(size_t) * hostptr->aik_size_; 
      size_t uik_nkj_size = sizeof(std::pair<size_t, size_t>) * hostptr->uik_nkj_size_; 
      size_t lij_ujk_size = sizeof(std::pair<size_t, size_t>) * hostptr->lij_ujk_size_;
      size_t do_aki_size = sizeof(char) * hostptr->do_aki_size_;
      size_t aki_size = sizeof(size_t) * hostptr->aki_size_;
      size_t lki_nkj_size = sizeof(std::pair<size_t, size_t>) * hostptr->lki_nkj_size_;
      size_t lkj_uji_size = sizeof(std::pair<size_t, size_t>) * hostptr->lkj_uji_size_;
      size_t uii_size = sizeof(size_t) * hostptr->uii_size_;

      /// Can not use "hipMalloc((void**)&devptr, sizeof(LuDecomposeConstDevice))" 
      /// because host variable "devptr" will contain addresss in the device memory (which is ok), 
      /// and "devptr->d_niLU_" becomes illegal since we can not access the address 
      /// in the device memory from the host code directly; see more discussion from:
      /// https://forums.developer.nvidia.com/t/cudamalloc-and-structs-and-pointers-problem/12266/2

      /// The solution is to keep devptr and its members as host variables;
      /// but its members can contain the addresses in the device memory.
      devptr = new LuDecomposeConstDevice;
      hipMalloc(&(devptr->d_niLU_),     niLU_size);
      hipMalloc(&(devptr->d_do_aik_),   do_aik_size);
      hipMalloc(&(devptr->d_aik_),      aik_size);
      hipMalloc(&(devptr->d_uik_nkj_),  uik_nkj_size);      
      hipMalloc(&(devptr->d_lij_ujk_),  lij_ujk_size);
      hipMalloc(&(devptr->d_do_aki_),   do_aki_size);
      hipMalloc(&(devptr->d_aki_),      aki_size);
      hipMalloc(&(devptr->d_lki_nkj_),  lki_nkj_size);
      hipMalloc(&(devptr->d_lkj_uji_),  lkj_uji_size);
      hipMalloc(&(devptr->d_uii_),      uii_size);

      /// copy the data from host to device
      hipMemcpy(devptr->d_niLU_,    hostptr->niLU_,    niLU_size,    hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_do_aik_,  hostptr->do_aik_,  do_aik_size,  hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_aik_,     hostptr->aik_,     aik_size,     hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_uik_nkj_, hostptr->uik_nkj_, uik_nkj_size, hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_lij_ujk_, hostptr->lij_ujk_, lij_ujk_size, hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_do_aki_,  hostptr->do_aki_,  do_aki_size,  hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_aki_,     hostptr->aki_,     aki_size,     hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_lki_nkj_, hostptr->lki_nkj_, lki_nkj_size, hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_lkj_uji_, hostptr->lkj_uji_, lkj_uji_size, hipMemcpyHostToDevice);
      hipMemcpy(devptr->d_uii_,     hostptr->uii_,     uii_size,     hipMemcpyHostToDevice);
      devptr->d_niLU_size_ = hostptr->niLU_size_;
    }

    /// This is the function that deletes the constant data members
    /// of objects with the "CudaLuDecomposition" type on the device
    void FreeConstData(LuDecomposeConstDevice* devptr)
    {
      hipFree(devptr->d_niLU_);
      hipFree(devptr->d_do_aik_);
      hipFree(devptr->d_aik_);
      hipFree(devptr->d_uik_nkj_);      
      hipFree(devptr->d_lij_ujk_);
      hipFree(devptr->d_do_aki_);
      hipFree(devptr->d_aki_);
      hipFree(devptr->d_lki_nkj_);
      hipFree(devptr->d_lkj_uji_);
      hipFree(devptr->d_uii_);
    }

    std::chrono::nanoseconds DecomposeKernelDriver(CudaSparseMatrixParam& sparseMatrix, LuDecomposeConstDevice* devptr)
    {
      /// create device pointers and allocate device memory
      double* d_A;
      double* d_L;
      double* d_U;

      hipMalloc(&d_A, sizeof(double) * sparseMatrix.A_size_);
      hipMalloc(&d_L, sizeof(double) * sparseMatrix.L_size_);
      hipMalloc(&d_U, sizeof(double) * sparseMatrix.U_size_);

      /// copy data from host to device
      hipMemcpy(d_A, sparseMatrix.A_, sizeof(double) * sparseMatrix.A_size_, hipMemcpyHostToDevice);
      hipMemcpy(d_L, sparseMatrix.L_, sizeof(double) * sparseMatrix.L_size_, hipMemcpyHostToDevice);
      hipMemcpy(d_U, sparseMatrix.U_, sizeof(double) * sparseMatrix.U_size_, hipMemcpyHostToDevice);

      size_t num_block = (sparseMatrix.n_grids_ + BLOCK_SIZE - 1) / BLOCK_SIZE;

      std::cout << "num_block = " << num_block << ", BLOCK_SIZE = " << BLOCK_SIZE << std::endl;
      /// call CUDA kernel and measure the execution time
      auto startTime = std::chrono::high_resolution_clock::now();
      std::cout << "before the kernel..." << std::endl;  
      DecomposeKernel<<<num_block, BLOCK_SIZE>>>(d_A, d_L, d_U, devptr, sparseMatrix.n_grids_);
      std::cout << "after the kernel..." << std::endl; 
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      /// copy the data from device to host
      hipMemcpy(sparseMatrix.L_, d_L, sizeof(double) * sparseMatrix.L_size_, hipMemcpyDeviceToHost);
      hipMemcpy(sparseMatrix.U_, d_U, sizeof(double) * sparseMatrix.U_size_, hipMemcpyDeviceToHost);

      /// clean up
      hipFree(d_A);
      hipFree(d_L);
      hipFree(d_U);

      return kernel_duration;
    }  // end of DecomposeKernelDriver
  }    // end of namespace cuda
}      // end of namespace micm
