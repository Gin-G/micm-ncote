#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0
#include <chrono>
#include <iostream>
#include <micm/util/cuda_param.hpp>
#include <vector>
#include <micm/solver/cuda_lu_decomposition.hpp>

namespace micm
{
  namespace cuda
  {
    /// This is the CUDA kernel that performs LU decomposition on the device
    __global__ void DecomposeKernel(const double* d_A, double* d_L, double* d_U, 
                                    LuDecomposeConstDevice* devptr, size_t ngrids)
    {
      /// Local device variables
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      std::pair<size_t, size_t>* uik_nkj = devptr->d_uik_nkj_;
      std::pair<size_t, size_t>* lij_ujk = devptr->d_lij_ujk_;
      std::pair<size_t, size_t>* lkj_uji = devptr->d_lkj_uji_;
      std::pair<size_t, size_t>* lki_nkj = device->d_lki_nkj_;
      size_t niLu_size = devptr->d_niLU_.size();
      size_t do_aik_offset = 0;
      size_t aik_offset = 0;
      size_t uik_nkj_offset = 0;
      size_t lij_ujk_offset = 0;
      size_t do_aki_offset = 0;
      size_t aki_offset = 0;
      size_t lkj_uji_offset = 0;
      size_t lki_nkj_offset = 0;
      size_t uii_offset = 0;

      if (tid < ngrids)
      {
        // loop through every element in niLU
        for (size_t i = 0; i < niLU_size; i++)
        {
          // upper triangular matrix
          auto inLU = device->d_niLU_[i];
          for (size_t iU = 0; iU < inLU.second; ++iU)
          {
            if (device->d_do_aik_[do_aik_offset++])
            {
              size_t U_idx = uik_nkj[uik_nkj_offset].first + tid;
              size_t A_idx = device->d_aik_[aik_offset++] + tid;
              d_U[U_idx] = d_A[A_idx];
            }

            for (size_t ikj = 0; ikj < uik_nkj[uik_nkj_offset].second; ++ikj)
            {
              size_t U_idx_1 = uik_nkj[uik_nkj_offset].first + tid;
              size_t L_idx = lij_ujk[lij_ujk_offset].first + tid;
              size_t U_idx_2 = lij_ujk[lij_ujk_offset].second + tid;
              d_U[U_idx_1] -= d_L[L_idx] * d_U[U_idx_2];
              ++lij_ujk_offset;
            }
            ++uik_nkj_offset;
          }
          // lower triangular matrix

          d_L[lki_nkj[lki_nkj_offset++].first + tid] = 1.0;

          for (size_t iL = 0; iL < inLU.first; ++iL)
          {
            if (device->d_do_aki_[do_aki_offset++])
            {
              size_t L_idx = lki_nkj[lki_nkj_offset].first + tid;
              size_t A_idx = device->d_aki_[aki_offset++] + tid;
              d_L[L_idx] = d_A[A_idx];
            }
            for (size_t ikj = 0; ikj < lki_nkj[lki_nkj_offset].second; ++ikj)
            {
              size_t L_idx_1 = lki_nkj[lki_nkj_offset].first + tid;
              size_t L_idx_2 = lkj_uji[lkj_uji_offset].first + tid;
              size_t U_idx = lkj_uji[lkj_uji_offset].second + tid;
              d_L[L_idx_1] -= d_L[L_idx_2] * d_U[U_idx];
              ++lkj_uji_offset;
            }
            d_L[lki_nkj[lki_nkj_offset].first + tid] /= d_U[device->uii_[uii_offset] + tid];
            ++lki_nkj_offset;
            ++uii_offset;
          }
        }
      }
    }  // end of CUDA kernel

    /// This is the function that copies the constant data members
    /// of objects with the "CudaLuDecomposition" type to the device
    void CopyConstData(CudaLuDecomposition* self, LuDecomposeConstDevice* devptr)
    {
      /// allocate device memory for the device struct
      hipMalloc(&devptr,               sizeof(LuDecomposeConstDevice));
      hipMalloc(&(devptr->d_niLU_),    sizeof(std::pair<size_t, size_t>) * self.niLU_.size());
      hipMalloc(&(devptr->d_do_aik_),  sizeof(char) * self.do_aik_.size());
      hipMalloc(&(devptr->d_aik_),     sizeof(size_t) * self.aik_.size());
      hipMalloc(&(devptr->d_uik_nkj_), sizeof(std::pair<size_t, size_t>) * self.uik_nkj_.size());
      hipMalloc(&(devptr->d_lij_ujk_), sizeof(std::pair<size_t, size_t>) * self.lij_ujk_.size());
      hipMalloc(&(devptr->d_do_aki_),  sizeof(char) * self.do_aki_.size());
      hipMalloc(&(devptr->d_aki_),     sizeof(size_t) * self.aki_.size());
      hipMalloc(&(devptr->d_lki_nkj_), sizeof(std::pair<size_t, size_t>) * self.lki_nkj_.size());
      hipMalloc(&(devptr->d_lkj_uji_), sizeof(std::pair<size_t, size_t>) * self.lkj_uji_.size());
      hipMalloc(&(devptr->d_uii_),     sizeof(size_t) * self.uii_.size());

      /// copy the data from host to device
      hipMemcpy(&(devptr->d_niLU_),    self.niLU_.data(), sizeof(std::pair<size_t, size_t>) * self.niLU_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_do_aik_),  self.do_aik_.data(), sizeof(char) * self.do_aik_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_aik_),     self.aik_.data(), sizeof(size_t) * self.aik_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_uik_nkj_), self.uik_nkj_.data(), sizeof(std::pair<size_t, size_t>) * self.uik_nkj_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_lij_ujk_), self.lij_ujk_.data(), sizeof(std::pair<size_t, size_t>) * self.lij_ujk_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_do_aki_),  self.do_aki_.data(), sizeof(char) * self.do_aki_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_aki_),     self.aki_.data(), sizeof(size_t) * self.aki_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_lki_nkj_), self.lki_nkj_.data(), sizeof(std::pair<size_t, size_t>) * self.lki_nkj_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_lkj_uji_), self.lkj_uji_.data(), sizeof(std::pair<size_t, size_t>) * self.lkj_uji_.size(), hipMemcpyHostToDevice);
      hipMemcpy(&(devptr->d_uii_),     self.uii_.data(), sizeof(size_t) * self.uii_.size(), hipMemcpyHostToDevice);
    }

    /// This is the function that deletes the constant data members
    /// of objects with the "CudaLuDecomposition" type on the device
    void FreeConstData(LuDecomposeConstDevice* devptr)
    {
      hipFree(devptr);
    }

    std::chrono::nanoseconds DecomposeKernelDriver(CudaSparseMatrixParam& sparseMatrix, LuDecomposeConstDevice* devptr)
    {
      /// create device pointers and allocate device memory
      double* d_A;
      double* d_L;
      double* d_U;

      hipMalloc(&d_A, sizeof(double) * sparseMatrix.A_size_);
      hipMalloc(&d_L, sizeof(double) * sparseMatrix.L_size_);
      hipMalloc(&d_U, sizeof(double) * sparseMatrix.U_size_);

      /// copy data from host to device
      hipMemcpy(d_A, sparseMatrix.A_, sizeof(double) * sparseMatrix.A_size_, hipMemcpyHostToDevice);
      hipMemcpy(d_L, sparseMatrix.L_, sizeof(double) * sparseMatrix.L_size_, hipMemcpyHostToDevice);
      hipMemcpy(d_U, sparseMatrix.U_, sizeof(double) * sparseMatrix.U_size_, hipMemcpyHostToDevice);

      size_t num_block = (sparseMatrix.n_grids_ + BLOCK_SIZE - 1) / BLOCK_SIZE;

      /// call CUDA kernel and measure the execution time
      auto startTime = std::chrono::high_resolution_clock::now();
      DecomposeKernel<<<num_block, BLOCK_SIZE>>>(d_A, d_L, d_U, devptr, sparseMatrix.n_grids_);
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      /// copy the data from device to host
      hipMemcpy(sparseMatrix.L_, d_L, sizeof(double) * sparseMatrix.L_size_, hipMemcpyDeviceToHost);
      hipMemcpy(sparseMatrix.U_, d_U, sizeof(double) * sparseMatrix.U_size_, hipMemcpyDeviceToHost);
      
      /// clean up
      hipFree(d_A);
      hipFree(d_L);
      hipFree(d_U);

      return kernel_duration;
    }  // end of DecomposeKernelDriver
  }    // end of namespace cuda
}      // end of namespace micm
