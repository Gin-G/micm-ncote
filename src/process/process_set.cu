#include "hip/hip_runtime.h"
#include <iostream>
#include <micm/solver/state.hpp>
#include <micm/util/matrix.hpp>
#include <micm/util/sparse_matrix.hpp>

namespace micm
{

  // one thread per reaction
  // passing all device pointers
  __global__ void AddForcingTerms_kernel(
      double* rate_constants,
      double* state_variables,
      double* forcing,
      int matrix_rows,
      int rate_constants_columns,
      int state_forcing_columns,
      size_t* number_of_reactants_,
      size_t* accumulated_n_reactants,
      size_t* reactant_ids_,
      size_t* number_of_products_,
      size_t* accumulated_n_products,
      size_t* product_ids_,
      double* yields_)
  {
    // define thread index
    int tid = blockIdx.x + blockDim.x + threadIdx.x;
    int rate_reactants_size = matrix_rows * rate_constants_columns;
    if (tid < rate_reactants_size)
    {
      double rate = rate_constants[tid];  // rate of a specific reaction in a specific gridcell
      int row_index = tid % rate_constants_columns;
      int reactant_num = number_of_reactants_[tid % rate_constants_columns];  // number of reactants of the reaction
      int product_num = number_of_products_[tid % rate_constants_columns];    // number of products of the reaction
      int initial_reactant_ids_index = accumulated_n_reactants[tid % rate_constants_columns];
      int initial_product_ids_index = accumulated_n_products[tid % rate_constants_columns];
      int initial_yields_index = accumulated_n_products[tid % rate_constants_columns];

      // access index at reactant_ids based on number_of_reactant_
      for (int i_reactant = 0; i_reactant < reactant_num; i_reactant++)
      {
        int reactant_ids_index = i_reactant + initial_reactant_ids_index;
        int state_forcing_col_index = reactant_ids_[reactant_ids_index];
        // how to match thread idx to state_variable index
        // but we need to consider the row of state_variable
        rate *= state_variables[row_index * state_forcing_columns + state_forcing_col_index];
      }

      for (int i_reactant = 0; i_reactant < reactant_num; i_reactant++)
      {
        int reactant_ids_index = i_reactant + initial_reactant_ids_index;
        int state_forcing_col_index = reactant_ids_[reactant_ids_index];

        forcing[row_index * state_forcing_columns + state_forcing_col_index] -= rate;
      }

      for (int i_product = 0; i_product < product_num; i_product++)
      {
        int yields_index = initial_yields_index + i_product;
        int product_ids_index = initial_product_ids_index + i_product;
        int forcing_col_index = product_ids_[product_ids_index];
        forcing[row_index * state_forcing_columns + forcing_col_index] += yields_[yields_index] * rate;
      }
    }
  }

  void AddForcingTerms_kernelSetup(
      const Matrix<double>& rate_constants,
      const Matrix<double>& state_variables,
      Matrix<double>& forcing,
      const std::vector<std::size_t>& number_of_reactants_,
      const std::vector<std::size_t>& reactant_ids_,
      const std::vector<std::size_t>& number_of_products_,
      const std::vector<std::size_t>& product_ids_,
      const std::vector<double>& yields_,
      const std::vector<std::size_t>& jacobian_flat_ids_)
  {
    int matrix_rows = rate_constants.size();
    int rate_constants_columns = rate_constants[0].size();
    int state_forcing_columns = state_variables[0].size();
    // access class member vectors
    const size_t* number_of_reactants = number_of_reactants_.data();
    int number_of_reactants_size = number_of_reactants_.size();
    const size_t* reactant_ids = reactant_ids_.data();
    int reactant_ids_size = reactant_ids_.size();
    const size_t* number_of_products = number_of_products_.data();
    int number_of_products_size = number_of_products_.size();
    const size_t* product_ids = product_ids_.data();
    int product_ids_size = product_ids_.size();
    const double* yields = yields_.data();
    int yields_size = yields_.size();

    // allocate memory for host c pointers
    int accumulated_n_reactants_bytes = sizeof(size_t) * (number_of_reactants_size + 1);
    size_t* accumulated_n_reactants = (size_t*)malloc(accumulated_n_reactants_bytes);
    accumulated_n_reactants[0] = 0;
    for (int i = 0; i < number_of_reactants_size; i++)
    {
      int sum = accumulated_n_reactants[i] + number_of_reactants[i];
      accumulated_n_reactants[i + 1] = sum;
    }

    int accumulated_n_products_bytes = sizeof(size_t) * (number_of_products_size + 1);
    size_t* accumulated_n_products = (size_t*)malloc(accumulated_n_products_bytes);
    accumulated_n_products[0] = 0;
    for (int i = 0; i < number_of_products_size; i++)
    {
      int sum = accumulated_n_products[i] + number_of_products[i];
      accumulated_n_products[i + 1] = sum;
    }

    // device pointer to vectors
    double* d_rate_constants;
    double* d_state_variables;
    double* d_forcing;
    size_t* d_number_of_reactants_;
    size_t* d_accumulated_n_reactants;
    size_t* d_reactant_ids_;
    size_t* d_number_of_products_;
    size_t* d_accumulated_n_products;
    size_t* d_product_ids_;
    double* d_yields_;

    // allocate device memory
    size_t rate_constants_bytes = sizeof(double) * (matrix_rows * rate_constants_columns);
    size_t state_variables_bytes = sizeof(double) * (matrix_rows * state_forcing_columns);
    size_t forcing_bytes = sizeof(double) * (matrix_rows * state_forcing_columns);
    size_t number_of_reactants_bytes = sizeof(size_t) * number_of_reactants_size;
    size_t reactant_ids_bytes = sizeof(size_t) * reactant_ids_size;
    size_t number_of_products_bytes = sizeof(size_t) * number_of_products_size;
    size_t product_ids_bytes = sizeof(size_t) * product_ids_size;
    size_t yields_bytes = sizeof(double) * yields_size;

    hipMalloc(&d_rate_constants, rate_constants_bytes);
    hipMalloc(&d_state_variables, state_variables_bytes);
    hipMalloc(&d_forcing, forcing_bytes);
    hipMalloc(&d_number_of_reactants_, number_of_reactants_bytes);
    hipMalloc(&d_accumulated_n_reactants, accumulated_n_reactants_bytes);
    hipMalloc(&d_reactant_ids_, reactant_ids_bytes);
    hipMalloc(&d_number_of_products_, number_of_products_bytes);
    hipMalloc(&d_accumulated_n_products, accumulated_n_products_bytes);
    hipMalloc(&d_product_ids_, product_ids_bytes);
    hipMalloc(&d_yields_, yields_bytes);

    // copy data from host to device memory
    const double* rate_constants_data = rate_constants.AsVector().data();
    const double* state_variables_data = state_variables.AsVector().data();
    double* forcing_data = forcing.AsVector().data();

    hipMemcpy(d_rate_constants, rate_constants_data, rate_constants_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_state_variables, state_variables_data, state_variables_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_forcing, forcing_data, forcing_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_reactants_, number_of_reactants, number_of_reactants_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_accumulated_n_reactants, accumulated_n_reactants, accumulated_n_reactants_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_reactant_ids_, reactant_ids, reactant_ids_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_products_, number_of_products, number_of_products_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_accumulated_n_products, accumulated_n_products, accumulated_n_products_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_product_ids_, product_ids, product_ids_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_yields_, yields, yields_bytes, hipMemcpyHostToDevice);

    // total thread count == rate_constants matrix size?
    int threads_count = matrix_rows * rate_constants_columns;
    // block size
    int threadsPerBlock = 128;  // 32 threads per warp * 4 warps
    // grid size
    int blocks_count = (int)ceil(threads_count / threadsPerBlock);

    // kernel function call
    AddForcingTerms_kernel<<<blocks_count, threadsPerBlock>>>(
        d_rate_constants,
        d_state_variables,
        d_forcing,
        matrix_rows,
        rate_constants_columns,
        state_forcing_columns,
        d_number_of_reactants_,
        d_accumulated_n_reactants,
        d_reactant_ids_,
        d_number_of_products_,
        d_accumulated_n_products,
        d_product_ids_,
        d_yields_);
    hipDeviceSynchronize();
    hipMemcpy(d_forcing, forcing_data, forcing_bytes, hipMemcpyDeviceToHost);

    hipFree(d_rate_constants);
    hipFree(d_state_variables);
    hipFree(d_forcing);
    hipFree(d_number_of_reactants_);
    hipFree(d_accumulated_n_reactants);
    hipFree(d_reactant_ids_);
    hipFree(d_number_of_products_);
    hipFree(d_accumulated_n_products);
    hipFree(d_product_ids_);
    hipFree(d_yields_);

    free(accumulated_n_reactants);
    free(accumulated_n_products);
  }
}  // namespace micm