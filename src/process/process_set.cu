
#include <hip/hip_runtime.h>
#include <iostream>

namespace micm {
    namespace cuda {
      // flipped memory layout
    __global__ void AddForcingTerms_kernel(
        double* rate_constants, 
        double* state_variables, 
        double* forcing, 
        int ngrids,
        int nrxns, 
        int nspecs, 
        size_t* number_of_reactants_, 
        size_t* reactant_ids_, 
        size_t* number_of_products_, 
        size_t* product_ids_, 
        double* yields_)
    {
      //define thread index 
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x; 
      size_t react_id_offset, prod_id_offset, yield_offset;

      if (tid < ngrids){
         react_id_offset = 0;
	 prod_id_offset = 0;
	 yield_offset = 0;
         for (std::size_t i_rxn = 0; i_rxn < nrxns; ++i_rxn)
         {
           double rate = rate_constants[i_rxn*ngrids+tid];
           size_t index;

           for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react)
             rate *= state_variables[reactant_ids_[react_id_offset+i_react]*ngrids+tid];
           for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react){
             forcing[reactant_ids_[react_id_offset+i_react]*ngrids+tid] -= rate;
	   }
           for (std::size_t i_prod = 0; i_prod < number_of_products_[i_rxn]; ++i_prod){
             index = product_ids_[prod_id_offset+i_prod]*ngrids+tid;
             forcing[index] += yields_[yield_offset+i_prod] * rate;
           }
           react_id_offset += number_of_reactants_[i_rxn];
           prod_id_offset += number_of_products_[i_rxn];
           yield_offset += number_of_products_[i_rxn];
         } // for loop over number of reactions
      }    // if check for valid CUDA threads
    }      // end of AddForcingTerms_kernel


__global__ void AddJacobianTerms_kernel(
  double* rate_constants,
  double* state_variables,
  double* jacobian,
  int n_grids,
  int n_reactions,
  int n_species,
  size_t*  number_of_reactants,
  size_t* reactant_ids,
  size_t* number_of_products,
  size_t* product_ids,
  double* yields,
  size_t* jacobian_flat_ids,
  size_t rows_ids_size){
  
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 
  size_t react_id_offset, prod_id_offset, yield_offset, jacobian_id_idx, initial_jacobian_idx; 
   
    //paralle grids -> one thread per grid cell
    //how to get jacobian initial index for each grid
    if (tid < n_grids){
      printf("running in kernel\n"); 
      react_id_offset = 0, prod_id_offset = 0, yield_offset = 0, jacobian_id_idx = -1; 
      size_t* jacobian_flat_ids_ptr = jacobian_flat_ids; 
      initial_jacobian_idx = tid * rows_ids_size; 
      for (int i_rxn = 0; i_rxn < n_reactions; i_rxn++){
          printf ("reaction index %d\n",i_rxn); 
          for(int i_ind = 0; i_ind < number_of_reactants[i_rxn]; i_ind++){
             double d_rate_d_int = rate_constants[i_rxn * n_grids + tid]; 
             for (int i_react = 0; i_react < number_of_reactants[i_rxn]; i_react++){
              if (i_react != i_ind){
                d_rate_d_int *= state_variables[reactant_ids[react_id_offset + i_react] * n_grids + tid];
              }
             }//second inner loop
       }//first inner loop
      }//loop over num_reactions
    }//check for valid tid 
  }// end of AddJacobianTerms_kernel
    

    void AddForcingTerms_kernelSetup(
        const double* rate_constants_data,
        const double* state_variables_data,
        double* forcing_data,
        int ngrids,
        int nrxns,
        int nspecs,
        const size_t* number_of_reactants,
        int number_of_reactants_size,
        const size_t* reactant_ids, 
        int reactant_ids_size,
        const size_t* number_of_products, 
        int number_of_products_size,
        const size_t* product_ids,
        int product_ids_size,
        const double* yields,
        int yields_size){
       
        // device pointer to vectorss
        double* d_rate_constants; 
        double* d_state_variables; 
        double* d_forcing; 
        double* d_yields_; 
        size_t* d_number_of_reactants_; 
        size_t* d_reactant_ids_; 
        size_t* d_number_of_products_; 
        size_t* d_product_ids_; 
       
        //allocate device memory
        size_t rate_constants_bytes = sizeof(double) * (ngrids * nrxns); 
        size_t state_forcing_bytes = sizeof(double) * (ngrids * nspecs); 
        size_t yields_bytes = sizeof(double) * yields_size;
        size_t number_of_reactants_bytes = sizeof(size_t) * number_of_reactants_size;
        size_t reactant_ids_bytes = sizeof(size_t) * reactant_ids_size; 
        size_t number_of_products_bytes = sizeof(size_t) * number_of_products_size; 
        size_t product_ids_bytes = sizeof(size_t) * product_ids_size;
        
        hipMalloc(&d_rate_constants, rate_constants_bytes); 
        hipMalloc(&d_state_variables, state_forcing_bytes); 
        hipMalloc(&d_forcing, state_forcing_bytes); 
        hipMalloc(&d_number_of_reactants_, number_of_reactants_bytes);
        hipMalloc(&d_reactant_ids_, reactant_ids_bytes);  
        hipMalloc(&d_number_of_products_, number_of_products_bytes);
        hipMalloc(&d_product_ids_, product_ids_bytes);  
        hipMalloc(&d_yields_, yields_bytes); 

        //copy data from host memory to device memory    
        hipMemcpy(d_rate_constants, rate_constants_data, rate_constants_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_state_variables, state_variables_data, state_forcing_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_forcing, forcing_data, state_forcing_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_reactants_, number_of_reactants, number_of_reactants_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_reactant_ids_, reactant_ids, reactant_ids_bytes,hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_products_, number_of_products, number_of_products_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_product_ids_, product_ids, product_ids_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_yields_, yields, yields_bytes, hipMemcpyHostToDevice); 

        //total thread count == number of grid cells
        int block_size = 32; 
        int num_block = (ngrids + block_size - 1)/block_size; 
        
        //kernel function call
        AddForcingTerms_kernel<<<num_block, block_size>>>(
            d_rate_constants, 
            d_state_variables, 
            d_forcing, 
            ngrids, nrxns, nspecs,
            d_number_of_reactants_, 
            d_reactant_ids_, 
            d_number_of_products_, 
            d_product_ids_, 
            d_yields_);
        hipDeviceSynchronize(); 
        
        // copy data from device memory to host memory
        hipMemcpy(forcing_data, d_forcing, state_forcing_bytes, hipMemcpyDeviceToHost);
   
        // clean up	
        hipFree(d_rate_constants); 
        hipFree(d_state_variables); 
        hipFree(d_forcing);
        hipFree(d_number_of_reactants_);
        hipFree(d_reactant_ids_);
        hipFree(d_number_of_products_);
        hipFree(d_product_ids_);
        hipFree(d_yields_);
    } // end of AddForcingTerms_kernelSetup
    

    void AddJacobianTerms_kernelSetup(
        const double* rate_constants,
        const double* state_variables,
        double* jacobian,
        size_t n_grids,
        size_t n_reactions,
        size_t n_species,
        size_t jacobian_size,
        size_t row_ids_size,
        const size_t* number_of_reactants, 
        const size_t* reactant_ids, 
        size_t reactant_ids_size, 
        const size_t* number_of_products, 
        const size_t* product_ids,
        size_t product_ids_size, 
        const double* yields,
        size_t yields_size,
        size_t* jacobian_flat_ids,
        size_t jacobian_flat_ids_size){
        std::cout << "grid size: "<< n_grids<<std::endl; 
        std::cout << "reaction size: "<< n_reactions<<std::endl; 
        std::cout << "species size: "<<n_species<<std::endl; 
        //device pointer
        double* d_rate_constants; 
        double* d_state_variables; 
        double* d_jacobian; 
        size_t* d_number_of_reactants;    
        size_t* d_reactant_ids; 
        size_t* d_number_of_products; 
        size_t* d_product_ids; 
        double* d_yields; 
        size_t* d_jacobian_flat_ids; 

        //allocate device memory 
        hipMalloc(&d_rate_constants, sizeof(double)* n_grids * n_reactions); 
        hipMalloc(&d_state_variables, sizeof(double)* n_grids * n_species); 
        hipMalloc(&d_jacobian, sizeof(double)* jacobian_size);
        hipMalloc(&d_number_of_reactants, sizeof(size_t)* n_reactions); 
        hipMalloc(&d_reactant_ids, sizeof(size_t) * reactant_ids_size);
        hipMalloc(&d_number_of_products, sizeof(size_t)* n_reactions);
        hipMalloc(&d_product_ids, sizeof(size_t) * product_ids_size);  
        hipMalloc(&yields, sizeof(double) * yields_size); 
        hipMalloc(&jacobian_flat_ids, sizeof(size_t)* jacobian_flat_ids_size); 


        //transfer data from host to device 
        hipMemcpy(d_rate_constants, rate_constants, sizeof(double)* n_grids * n_reactions,hipMemcpyHostToDevice); 
        hipMemcpy(d_state_variables, state_variables, sizeof(double)* n_grids * n_species, hipMemcpyHostToDevice); 
        hipMemcpy(d_jacobian, jacobian, sizeof(double)* jacobian_size, hipMemcpyHostToDevice);
        hipMemcpy(d_number_of_reactants, number_of_reactants, sizeof(size_t)* n_reactions,hipMemcpyHostToDevice); 
        hipMemcpy(d_reactant_ids, reactant_ids, sizeof(size_t) * reactant_ids_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_products, number_of_products, sizeof(size_t)* n_reactions, hipMemcpyHostToDevice); 
        hipMemcpy(d_product_ids, product_ids, sizeof(size_t)* product_ids_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_yields, yields, sizeof(double) * yields_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_jacobian_flat_ids, jacobian_flat_ids, sizeof(size_t)* jacobian_flat_ids_size, hipMemcpyHostToDevice); 

        //total thread count == n_grids 
        int block_size = 32; 
        int num_blocks = (n_grids + block_size -1)/block_size; 
        //kernel function call
        AddJacobianTerms_kernel<<<num_blocks, block_size>>>(
          d_rate_constants,
          d_state_variables,
          d_jacobian,
          n_grids,
          n_reactions,
          n_species,
          d_number_of_reactants, 
          d_reactant_ids, 
          d_number_of_products, 
          d_product_ids,
          d_yields,
          d_jacobian_flat_ids,
          row_ids_size
        );
          hipDeviceSynchronize(); 
          hipMemcpy(jacobian, d_jacobian, sizeof(double)* jacobian_size, hipMemcpyDeviceToHost);
        
        // clean up	
        hipFree(d_rate_constants); 
        hipFree(d_state_variables); 
        hipFree(d_jacobian);
        hipFree(d_number_of_reactants);
        hipFree(d_reactant_ids);
        hipFree(d_number_of_products);
        hipFree(d_product_ids);
        hipFree(d_yields);
        hipFree(d_jacobian_flat_ids); 
    }//end of AddJacobianTerms_kernelSetup function 
  } // namespace cuda 
}     // namespace micm
