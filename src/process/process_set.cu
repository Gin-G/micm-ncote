#include "hip/hip_runtime.h"
#include <micm/process/process.hpp>
#include <micm/solver/state.hpp>
#include <micm/util/matrix.hpp>
#include <micm/util/sparse_matrix.hpp>
#include <iostream>

namespace micm {

    void AddForcingTerms_kernelSetup(
        const Matrix<double>& rate_constants,
        const Matrix<double>& state_variables,
        Matrix<double>& forcing, size_t* number_of_reactants_, int number_of_reactants_size, 
        size_t* reactant_ids_, int reactant_ids_size, size_t* number_of_products_, int number_of_products_size,
        size_t* product_ids_, int product_ids_size, size_t* yields_, int yields_size) const
    {
        
        //allocate memory for host c pointers 
        int accumulated_n_reactants_bytes = sizeof(size_t) * (number_of_reactants_size + 1); 
        size_t* accumulated_n_reactants = (size_t*)malloc(accumulated_n_reactants_bytes); 
        accumulated_n_reactants[0] = 0; 
        for (int i = 0; i < number_of_reactants_size; i++){
            int sum = accumulated_n_reactants[i] + number_of_reactants_[i]; 
            accumulated_n_reactants[i+1] = sum; 
        }    
        
        int accumulated_n_products_bytes = sizeof(size_t) * (number_of_products_size + 1); 
        size_t* accumulated_n_products = (size_t*)malloc(accumulated_n_products_bytes); 
        accumulated_n_products[0] = 0;  
        for (int i = 0; i < number_of_products_size; i++){
            int sum = accumulated_n_products[i] + number_of_products_[i]; 
            accumulated_n_products[i+1] = sum; 
        }

        // device pointer to vectors
        std::double* d_rate_constants; 
        std::double* d_state_variables; 
        std::double* d_forcing; 
        std::size_t* d_number_of_reactants_; 
        std::size_t* d_accumulated_n_reactants; 
        std::size_t* d_reactant_ids_; 
        std::size_t* d_number_of_products_; 
        std::size_t* d_accumulated_n_products; 
        std::size_t* d_product_ids_; 
        std::size_t* d_yields_; 
        
        size_t rate_constants_bytes = sizeof(std::double) * rate_constants.data_.size(); 
        size_t state_variables_bytes = sizeof(std::double) * state_variables.data_.size();
        size_t forcing_bytes = sizeof(std::double)* forcing.data_.size(); 
        size_t number_of_reactants_bytes = sizeof(std::size_t) * number_of_reactants_size;
        size_t reactant_ids_bytes = sizeof(std::size_t) * reactant_ids_size; 
        size_t number_of_products_bytes = sizeof(std::size_t) * number_of_products_size; 
        size_t product_ids_bytes = sizeof(std::size_t) * product_ids_size;
        size_t yields_bytes = sizeof(std::size_t) * yields_size;
        
        //allocate device memory
        hipMalloc(&d_rate_constants, rate_constants_bytes); 
        hipMalloc(&d_state_variables, state_variables_bytes); 
        hipMalloc(&d_forcing, forcing_bytes); 
        hipMalloc(&d_number_of_reactants_, number_of_reactants_bytes);
        hipMalloc(&d_accumulated_n_reactants, accumulated_n_reactants_bytes); 
        hipMalloc(&d_reactant_ids_, reactant_ids_bytes);  
        hipMalloc(&d_number_of_products_, number_of_products_bytes);
        hipMalloc(&d_accumulated_n_products, accumulated_n_products_bytes); 
        hipMalloc(&d_product_ids_, product_ids_bytes);  
        hipMalloc(&d_yields, yields_bytes); 
        
        //copy data from host to device memory 
        hipMemcpy(d_rate_constants, &rate_constants.data_, rate_constants_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_state_variables, &state_variables.data_, state_variables_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_forcing, &forcing.data_, forcing_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_reactants_, number_of_reactants_, number_of_reactants_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_accumulated_n_reactants, accumulated_n_reactants, accumulated_n_reactants_bytes,hipMemcpyHostToDevice); 
        hipMemcpy(d_reactant_ids_, reactant_ids_, reactant_ids_bytes,hipMemcpyHostToDevice); 
        hipMemcpy(d_number_of_products_, number_of_products_, number_of_products_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_accumulated_n_products, accumulated_n_products, accumulated_n_products_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_product_ids_, product_ids_, product_ids_bytes, hipMemcpyHostToDevice); 
        hipMemcpy(d_yields, yields_, yields_bytes, hipMemcpyHostToDevice); 

        //total thread count == rate_constants matrix size?
        int threads_count = rate_constants.x_dim * rate_constants.y_dim;
        //block size 
        int threadsPerBlock = 128; //32 threads per warp * 4 warps
        //grid size 
        int blocks_count = (int)ceil(threads_count/threadsPerBlock); 

        int matrix_rows = rate_constants.x_dim_; 
        int rate_constants_columns = rate_constants.y_dim_; 
        int state_forcing_columns = state_variables.y_dim_;

        //kernel function call
        AddForcingTerms_kernel(d_rate_constants, d_state_variables, 
        d_forcing, matrix_rows, rate_constants_columns, state_forcing_columns, 
        number_of_reactants_, accumulated_n_reactants, reactant_ids_, 
        number_of_products_, accumulated_n_products, product_ids_, 
        yields_);
        hipDeviceSynchronize(); 
        
        hipMemcpy(d_forcing, &forcing.data, hipMemcpyDeviceToHost);

        hipFree(d_rate_constants); 
        hipFree(d_state_variables); 
        hipFree(d_forcing);
        hipFree(d_number_of_reactants_);
        hipFree(d_accumulated_n_reactants);
        hipFree(d_reactant_ids_);
        hipFree(d_number_of_products_);
        hipFree(d_accumulated_n_products);
        hipFree(d_product_ids_);
        hipFree(d_yields_ );

    }


    //one thread per reaction in atompheric model 
    __global__ void AddForcingTerms_kernel(std::double* rate_constants, int rate_reactants_size, std::double* state_variables, 
    std::double* forcing, int matrix_rows, int rate_constants_columns, int state_forcing_columns, 
    std::size_t* number_of_reactants_, std::size_t* accumulated_n_reactants, std:: size_t* reactant_ids_, 
    std::size_t* number_of_products_, std::size_t* accumulated_n_products, std::size_t* product_ids_, 
    std::size_t* yields_){

    //define thread index 
    int tid = blockIdx.x + blockDim.x + threadIdx.x; 

    if (tid < rate_reactants_size){
        int rate = rate_constants[tid]; // rate of a specific reaction in a specific gridcell 
        int row_index = tid % rate_constants_columns; 
        int reactant_num = number_of_reactants_[tid % rate_constants_columns]; //number of reactants of the reaction
        int product_num = number_of_products_[tid % rate_constants_columns]; //number of products of the reaction 
        int initial_reactant_ids_index = accumulated_n_reactants[tid % rate_constants_columns];
        int initial_product_ids_index = accumulated_n_products[tid % rate_constants_columns];
        int initial_yields_index = accumulated_n_products[tid % rate_constants_columns]; 
        
        //access index at reactant_ids based on number_of_reactant_
        for (int i_reactant = 0; i_reactant < reactant_num; i_reactant++){
            int reactant_ids_index = i_reactant + initial_reactant_ids_index; 
            int state_forcing_col_index = reactant_ids_[reactant_ids_index]; 
            //how to match thread idx to state_variable index 
            //but we need to consider the row of state_variable 
            rate *= state_variables[row_index * state_forcing_columns + state_forcing_col_index]; 
            forcing[row_index * state_forcing_columns + state_forcing_col_index] -= rate; 
        }
        for (int i_product = 0; i_product < product_num; i_product++){
            int yields_index = initial_yields_index + i_product; 
            int product_ids_index  = initial_product_ids_index + i_product; 
            int forcing_col_index = product_ids_[product_ids_index]; 
            forcing[row_index * state_forcing_columns + forcing_col_index] += yields_[yields_index] * rate; 
        }   
    }
    }
}